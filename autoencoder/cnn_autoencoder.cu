#include "cnn_autoencoder.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


std::pair<float, float> denoise(const uint8_t* input, uint8_t* result, param* weights, int height, int width)
{
    // start timer
	hipEvent_t global_start, global_stop;
	hipEventCreate(&global_start);
	hipEventCreate(&global_stop); 
    float time = 0;
    hipEventRecord(global_start, 0);

    // preprocess image: uint8 [0, 255] -> float [0, 1] 
    uint8_t *dev_uint_input;
    float *dev_input;
    hipMalloc((void**)&dev_uint_input, height *width * sizeof(uint8_t));
    hipMalloc((void**)&dev_input, height *width * sizeof(float));
    hipMemcpy(dev_uint_input, input, height * width * sizeof(uint8_t), hipMemcpyHostToDevice);
    dim3 img_block_size(32, 32);
    dim3 img_grid_size(width/32+1, height/32+1);
    img2float<<<img_grid_size, img_block_size>>>(dev_uint_input, dev_input, height, width);
    // preprocessing done

    //*****ENCODER*****
    // - layer 1
    int out_channels = weights[0].dim_0;
    int in_channels = weights[0].dim_1;
    float *dev_pool_layer1;
    int layer1_out_height = height/2, layer1_out_width = width/2;
    hipMalloc((void**)&dev_pool_layer1, layer1_out_height * layer1_out_width * out_channels * sizeof(float));
    float layer1_time = encoder_layer(dev_input, dev_pool_layer1, weights[0].weight, weights[0].bias, in_channels, out_channels, height, width);
    // - layer 2
    out_channels = weights[1].dim_0;
    in_channels = weights[1].dim_1;
    float *dev_pool_layer2;
    int layer2_out_height = layer1_out_height/2, layer2_out_width = layer1_out_width/2;
    hipMalloc((void**)&dev_pool_layer2, layer2_out_height * layer2_out_width * out_channels * sizeof(float));
    float layer2_time = encoder_layer(dev_pool_layer1, dev_pool_layer2, weights[1].weight, weights[1].bias, in_channels, out_channels, layer1_out_height, layer1_out_width);
    //*****DECODER*****
    // - layer 3
    out_channels = weights[2].dim_1;
    in_channels = weights[2].dim_0;
    float *dev_trans_layer3;
    int layer3_out_height = layer1_out_height, layer3_out_width = layer1_out_width;
    hipMalloc((void**)&dev_trans_layer3, layer3_out_height * layer3_out_width * out_channels * sizeof(float));
    float layer3_time = decoder_layer(dev_pool_layer2, dev_trans_layer3, weights[2].weight, weights[2].bias, in_channels, out_channels, layer2_out_height, layer2_out_width);
    // - layer 4
    out_channels = weights[3].dim_1;
    in_channels = weights[3].dim_0;
    float *dev_trans_layer4;
    int layer4_out_height = height, layer4_out_width = width;
    hipMalloc((void**)&dev_trans_layer4, layer4_out_height * layer4_out_width * out_channels * sizeof(float));
    float layer4_time = decoder_layer(dev_trans_layer3, dev_trans_layer4, weights[3].weight, weights[3].bias, in_channels, out_channels, layer3_out_height, layer3_out_width);
    // - layer 5
    out_channels = weights[4].dim_0;
    in_channels = weights[4].dim_1;
    float *dev_result;
    hipMalloc((void**)&dev_result, height * width * out_channels * sizeof(float));
    float layer5_time = refine_layer(dev_trans_layer4, dev_result, weights[4].weight, weights[4].bias, in_channels, out_channels, layer4_out_height, layer4_out_width);
    // postprocessing
    img2uint<<<img_grid_size, img_block_size>>>(dev_result, dev_uint_input, height, width);
    hipMemcpy(result, dev_uint_input, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_uint_input);
    hipFree(dev_pool_layer1);
    hipFree(dev_pool_layer2);
    hipFree(dev_trans_layer3);
    hipFree(dev_trans_layer4);
    hipFree(dev_result);

    hipEventRecord(global_stop, 0);
	hipEventSynchronize(global_stop);
    hipEventElapsedTime(&time, global_start, global_stop);
    hipEventDestroy(global_start);
    hipEventDestroy(global_stop);
    
    return std::make_pair(time, layer1_time + layer2_time + layer3_time + layer4_time + layer5_time);
}


float encoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    // memory
    float *dev_padded;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    float *dev_conv;
    hipMalloc((void**)&dev_conv, height * width * out_channels * sizeof(float));

    // -- timer on
    // hipEvent_t pad_start, pad_stop;
	// hipEventCreate(&pad_start);
	// hipEventCreate(&pad_stop); 
    // float pad_time = 0;
    // hipEventRecord(pad_start, 0);
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<pad_grid_size, pad_block_size, 0, stream[0]>>>(dev_input, dev_padded, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    // -- timer off
    // hipDeviceSynchronize();
    // hipEventRecord(pad_stop, 0);
	// hipEventSynchronize(pad_stop);
    // hipEventElapsedTime(&pad_time, pad_start, pad_stop);
    
    // - convolution
    // -- load weights
    hipMemcpyAsync(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    // -- timer on
    // hipEvent_t conv_start, conv_stop;
	// hipEventCreate(&conv_start);
	// hipEventCreate(&conv_stop); 
    // float conv_time = 0;
    // hipEventRecord(conv_start, 0);
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(width/32+1, height/32+1, out_channels);
    SharedConv2DReLU<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_conv,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // -- relu
    //ReLU<<<conv_grid_size, conv_block_size>>>(dev_conv, out_channels, height, width);
    // -- timer off
    // hipDeviceSynchronize();
    // hipEventRecord(conv_stop, 0);
	// hipEventSynchronize(conv_stop);
    // hipEventElapsedTime(&conv_time, conv_start, conv_stop);

    // - maxpool
    // -- timer on
    // hipEvent_t pool_start, pool_stop;
	// hipEventCreate(&pool_start);
	// hipEventCreate(&pool_stop); 
    // float pool_time = 0;
    // hipEventRecord(pool_start, 0);
    dim3 pool_block_size(32, 32, 1);
    dim3 pool_grid_size((width/2)/32+1, (height/2)/32+1, out_channels);
    MaxPool2D<<<pool_grid_size, pool_block_size>>>
    (
        dev_conv, dev_output, out_channels, height, width, 2,2,2,2
    );
    // -- timer off
    // hipDeviceSynchronize();
    // hipEventRecord(pool_stop, 0);
	// hipEventSynchronize(pool_stop);
    // hipEventElapsedTime(&pool_time, pool_start, pool_stop);

    // hipEventDestroy(pad_start);
    // hipEventDestroy(pad_stop);
    // hipEventDestroy(conv_start);
    // hipEventDestroy(conv_stop);
    // hipEventDestroy(pool_start);
    // hipEventDestroy(pool_stop);

    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);
    hipFree(dev_conv);

    //return pad_time + conv_time + pool_time;
    return 0;
}


float decoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    // memory
    float *dev_upsampled;
    int up_height = height + height-1, up_width = width + width-1;
    hipMalloc((void**)&dev_upsampled, up_height * up_width * in_channels * sizeof(float));
    //
    float *dev_padded;
    int up_pad = 1, down_pad = 2, left_pad = 1, right_pad = 2;
    int padded_height= up_height+up_pad+down_pad, padded_width = up_width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    //
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    //
    float *dev_conv_weight_flip;
    hipMalloc((void**)&dev_conv_weight_flip, out_channels*in_channels*3*3 * sizeof(float));
    //
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));

    // hipEvent_t up_start, up_stop;
	// hipEventCreate(&up_start);
	// hipEventCreate(&up_stop); 
    // float up_time = 0;
    // hipEventRecord(up_start, 0);
    dim3 up_block_size(32, 32, 1);
    dim3 up_grid_size(up_width/32+1, up_height/32+1, in_channels);
    ChessUpsample2D<<<up_grid_size, up_block_size,0, stream[0]>>>(dev_input, dev_upsampled, in_channels, height, width);
    // hipDeviceSynchronize();
    // hipEventRecord(up_stop, 0);
	// hipEventSynchronize(up_stop);
    // hipEventElapsedTime(&up_time, up_start, up_stop);

    // hipEvent_t pad_start, pad_stop;
	// hipEventCreate(&pad_start);
	// hipEventCreate(&pad_stop); 
    // float pad_time = 0;
    // hipEventRecord(pad_start, 0);
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<pad_grid_size, pad_block_size,0, stream[0]>>>(dev_upsampled, dev_padded, in_channels, up_height, up_width, up_pad, down_pad, left_pad, right_pad);
    // hipDeviceSynchronize();
    // hipEventRecord(pad_stop, 0);
	// hipEventSynchronize(pad_stop);
    // hipEventElapsedTime(&pad_time, pad_start, pad_stop);
    // -- convolve
    // ---- load weights
    
    hipMemcpyAsync(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    
    // hipEvent_t trans_start, trans_stop;
	// hipEventCreate(&trans_start);
	// hipEventCreate(&trans_stop); 
    // float trans_time = 0;
    // hipEventRecord(trans_start, 0);
    dim3 weight_block_size(32, 32, 1);
    dim3 weight_grid_size(3/32+1, 3/32+1, out_channels*in_channels);
    TransposeKernel<<<weight_grid_size, weight_block_size, 0, stream[1]>>>(dev_conv_weight, dev_conv_weight_flip, out_channels, in_channels, 3, 3);
    // hipDeviceSynchronize();
    // hipEventRecord(trans_stop, 0);
	// hipEventSynchronize(trans_stop);
    // hipEventElapsedTime(&trans_time, trans_start, trans_stop);
    // --- load bias

    // ---- perform conv
    // hipEvent_t conv_start, conv_stop;
	// hipEventCreate(&conv_start);
	// hipEventCreate(&conv_stop); 
    // float conv_time = 0;
    // hipEventRecord(conv_start, 0);
    int conv_height = height*2, conv_width = width*2;
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(conv_width/32+1, conv_height/32+1, out_channels);
    SharedConv2DReLU<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight_flip, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // relu
    //ReLU<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, conv_height, conv_width);
    // hipDeviceSynchronize();
    // hipEventRecord(conv_stop, 0);
	// hipEventSynchronize(conv_stop);
    // hipEventElapsedTime(&conv_time, conv_start, conv_stop);

    // hipEventDestroy(up_start);
    // hipEventDestroy(up_stop);
    // hipEventDestroy(pad_start);
    // hipEventDestroy(pad_stop);
    // hipEventDestroy(trans_start);
    // hipEventDestroy(trans_stop);
    // hipEventDestroy(conv_start);
    // hipEventDestroy(conv_stop);
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);
    hipFree(dev_upsampled);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);
    hipFree(dev_conv_weight_flip);

    //return up_time + pad_time + trans_time + conv_time;
    return 0;
}


float refine_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    float *dev_padded;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));

    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    // hipEvent_t pad_start, pad_stop;
	// hipEventCreate(&pad_start);
	// hipEventCreate(&pad_stop); 
    // float pad_time = 0;
    // hipEventRecord(pad_start, 0);
    ZeroPad2D<<<pad_grid_size, pad_block_size, 0, stream[0]>>>(dev_input, dev_padded, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    // hipStreamSynchronize(stream[0]);
    // hipEventRecord(pad_stop, 0);
	// hipEventSynchronize(pad_stop);
    // hipEventElapsedTime(&pad_time, pad_start, pad_stop);

    // -- convolve
    // ---- load weights
    hipMemcpyAsync(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    // ---- perform conv
    // hipEvent_t conv_start, conv_stop;
	// hipEventCreate(&conv_start);
	// hipEventCreate(&conv_stop); 
    // float conv_time = 0;
    // hipEventRecord(conv_start, 0);
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(width/32+1, height/32+1, out_channels);
    SharedConv2DSigmoid<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    //Sigmoid<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, height, width);
    // hipStreamSynchronize(stream[1]);
    // hipEventRecord(conv_stop, 0);
	// hipEventSynchronize(conv_stop);
    // hipEventElapsedTime(&conv_time, conv_start, conv_stop);

    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);
    // hipEventDestroy(conv_start);
    // hipEventDestroy(conv_stop);
    // hipEventDestroy(pad_start);
    // hipEventDestroy(pad_stop);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);

    //return conv_time+pad_time;
    return 0;
}