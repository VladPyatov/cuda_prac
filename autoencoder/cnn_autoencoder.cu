#include "cnn_autoencoder.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


std::pair<float, float> denoise(const uint8_t* input, uint8_t* result, param* weights, int height, int width)
{
    // start timer
	hipEvent_t global_start, global_stop;
	hipEventCreate(&global_start);
	hipEventCreate(&global_stop); 
    float time = 0;
    hipEventRecord(global_start, 0);

    // preprocess image: uint8 [0, 255] -> float [0, 1] 
    uint8_t *dev_uint_input;
    float *dev_input;
    hipMalloc((void**)&dev_uint_input, height *width * sizeof(uint8_t));
    hipMalloc((void**)&dev_input, height *width * sizeof(float));
    hipMemcpy(dev_uint_input, input, height * width * sizeof(uint8_t), hipMemcpyHostToDevice);
    dim3 img_block_size(32, 32);
    dim3 img_grid_size(width/32+1, height/32+1);
    img2float<<<img_grid_size, img_block_size>>>(dev_uint_input, dev_input, height, width);
    // preprocessing done

    //*****ENCODER*****
    // - layer 1
    int out_channels = weights[0].dim_0;
    int in_channels = weights[0].dim_1;
    float *dev_pool_layer1;
    int layer1_out_height = height/2, layer1_out_width = width/2;
    hipMalloc((void**)&dev_pool_layer1, layer1_out_height * layer1_out_width * out_channels * sizeof(float));
    float layer1_time = encoder_layer(dev_input, dev_pool_layer1, weights[0].weight, weights[0].bias, in_channels, out_channels, height, width);
    // - layer 2
    out_channels = weights[1].dim_0;
    in_channels = weights[1].dim_1;
    float *dev_pool_layer2;
    int layer2_out_height = layer1_out_height/2, layer2_out_width = layer1_out_width/2;
    hipMalloc((void**)&dev_pool_layer2, layer2_out_height * layer2_out_width * out_channels * sizeof(float));
    float layer2_time = encoder_layer(dev_pool_layer1, dev_pool_layer2, weights[1].weight, weights[1].bias, in_channels, out_channels, layer1_out_height, layer1_out_width);
    //*****DECODER*****
    // - layer 3
    out_channels = weights[2].dim_1;
    in_channels = weights[2].dim_0;
    float *dev_trans_layer3;
    int layer3_out_height = layer1_out_height, layer3_out_width = layer1_out_width;
    hipMalloc((void**)&dev_trans_layer3, layer3_out_height * layer3_out_width * out_channels * sizeof(float));
    float layer3_time = decoder_layer(dev_pool_layer2, dev_trans_layer3, weights[2].weight, weights[2].bias, in_channels, out_channels, layer2_out_height, layer2_out_width);
    // - layer 4
    out_channels = weights[3].dim_1;
    in_channels = weights[3].dim_0;
    float *dev_trans_layer4;
    int layer4_out_height = height, layer4_out_width = width;
    hipMalloc((void**)&dev_trans_layer4, layer4_out_height * layer4_out_width * out_channels * sizeof(float));
    float layer4_time = decoder_layer(dev_trans_layer3, dev_trans_layer4, weights[3].weight, weights[3].bias, in_channels, out_channels, layer3_out_height, layer3_out_width);
    // - layer 5
    out_channels = weights[4].dim_0;
    in_channels = weights[4].dim_1;
    float *dev_result;
    hipMalloc((void**)&dev_result, height * width * out_channels * sizeof(float));
    float layer5_time = refine_layer(dev_trans_layer4, dev_result, weights[4].weight, weights[4].bias, in_channels, out_channels, layer4_out_height, layer4_out_width);
    // postprocessing
    img2uint<<<img_grid_size, img_block_size>>>(dev_result, dev_uint_input, height, width);
    hipMemcpy(result, dev_uint_input, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_uint_input);
    hipFree(dev_pool_layer1);
    hipFree(dev_pool_layer2);
    hipFree(dev_trans_layer3);
    hipFree(dev_trans_layer4);
    hipFree(dev_result);

    hipEventRecord(global_stop, 0);
	hipEventSynchronize(global_stop);
    hipEventElapsedTime(&time, global_start, global_stop);
    hipEventDestroy(global_start);
    hipEventDestroy(global_stop);
    // // begin experiments
    // float array[] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27};
    // int array_h = 3;
    // int array_w = 3;
    // int array_c = 3;
    // int up_pad = 1;
    // int down_pad = 1;
    // int left_pad = 1;
    // int right_pad = 1;
    // float padded_array[(array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c] = {0};
    // print_array(array, array_h, array_w, array_c);
    // //print_array(padded_array, array_h, array_w, array_c);

    // float *dev_array;
    // float *dev_padded_array;
    
    // hipMalloc((void**)&dev_array, array_h * array_w * array_c * sizeof(float));
    // hipMalloc((void**)&dev_padded_array, (array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c * sizeof(float));

    // hipMemcpy(dev_array, array, array_h * array_w * array_c * sizeof(float), hipMemcpyHostToDevice);
    // const dim3 exp_block_size(32, 32, 1);
    // const dim3 exp_grid_size((array_w + left_pad + right_pad)/32+1, (array_h + up_pad + down_pad)/32+1, array_c);
    // ZeroPad2D<<<exp_grid_size, exp_block_size>>>(dev_array, dev_padded_array, array_c, array_h, array_w, up_pad, down_pad, left_pad, right_pad);
    // hipMemcpy(padded_array, dev_padded_array, (array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c * sizeof(float), hipMemcpyDeviceToHost);
    // // convolution
    // int out_channels = 2;
    // float weight[] = {
    //     1,2,0,-1,1,3,2,1,0, 2,1,2,0,1,2,0,0,1, 1,1,0,2,1,2,0,1,0,
    //     1,1,2,0,3,0,1,2,3, -1,2,1,0,1,2,0,1,0, 1,1,0,2,1,0,2,2,2,
    // };
    // float bias[] = {1, 2};

    // float *dev_convolved_array;
    // float *dev_weight;
    // float *dev_bias;
    
    // hipMalloc((void**)&dev_convolved_array, array_h * array_w * out_channels * sizeof(float));
    // hipMalloc((void**)&dev_weight, 3 * 3 * 6 * sizeof(float));
    // hipMalloc((void**)&dev_bias, 2 * sizeof(float));
    // hipMemcpy(dev_weight, weight, 3 * 3 * 6 * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(dev_bias, bias, 2 * sizeof(float), hipMemcpyHostToDevice);
    // dim3 conv_block_size(32, 32, 1);
    // dim3 conv_grid_size(array_w/32+1, array_h/32+1, out_channels);
    // Conv2D<<<conv_grid_size, conv_block_size>>>(dev_padded_array, dev_weight, dev_bias, dev_convolved_array, 3, 2, array_h + up_pad + down_pad, array_w + left_pad + right_pad, 3, 3);
    // hipMemcpy(array, dev_convolved_array, array_h * array_w * out_channels * sizeof(float), hipMemcpyDeviceToHost);
    // // MaxPool2D
    // float *dev_pooled_array;
    // float pooled_array[2*2*3];
    // hipMalloc((void**)&dev_pooled_array, ((array_w + left_pad + right_pad)/2) * ((array_h + up_pad + down_pad)/2) * array_c * sizeof(float));
    // dim3 pool_block_size(32, 32, 1);
    // dim3 pool_grid_size((array_w + left_pad + right_pad)/2/32+1, (array_h + up_pad + down_pad)/2/32+1, array_c);
    // MaxPool2D<<<pool_grid_size, pool_block_size>>>(dev_padded_array, dev_pooled_array, array_c, array_h + up_pad + down_pad, array_w + left_pad + right_pad, 2,2,2,2);
    // hipMemcpy(pooled_array, dev_pooled_array, ((array_w + left_pad + right_pad)/2) * ((array_h + up_pad + down_pad)/2) * array_c * sizeof(float), hipMemcpyDeviceToHost);
    // //Upsample
    // // MaxPool2D
    // float *dev_up_array;
    // float up_array[2*5*5];
    // hipMalloc((void**)&dev_up_array, 5*5*2* sizeof(float));
    // dim3 up_block_size(32, 32, 1);
    // dim3 up_grid_size(5/32+1, 5/32+1, 2);
    // ChessUpsample2D<<<up_grid_size, up_block_size>>>(dev_convolved_array, dev_up_array, 2, 3, 3);
    // hipMemcpy(up_array, dev_up_array, 2*5*5 * sizeof(float), hipMemcpyDeviceToHost);
    // // Flip Weight
    // printf("####WEIGHT####\n");
    // print_array(weight, 3, 3, 3*2);
    // dim3 weight_block_size(32, 32, 1);
    // dim3 weight_grid_size(3/32+1, 3/32+1, 2);
    // FlipWeight2D<<<weight_grid_size, weight_block_size>>>(dev_weight, 2, 3, 3, 3);
    // hipMemcpy(weight, dev_weight, 3*3*2*3 * sizeof(float), hipMemcpyDeviceToHost);
    // printf("####F_WEIGHT####\n");
    // print_array(weight, 3, 3, 3*2);
    // //hipFree(dev_array);
    // //hipFree(dev_padded_array);
    
    // print_array(padded_array, array_h + up_pad + down_pad, array_w + left_pad + right_pad, array_c);
    // print_array(array, array_h, array_w, out_channels);
    // print_array(pooled_array, (array_h + up_pad + down_pad)/2, (array_w + left_pad + right_pad)/2, array_c);
    // print_array(up_array, 5, 5, 2);
    // // end experiment
    return std::make_pair(time, layer1_time + layer2_time + layer3_time + layer4_time + layer5_time);
}


float encoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    // hipStream_t s1, s2;
    // cudaStreaCreate(&s1);
    // hipStreamCreate(&s2);
    // kernel1<<<...,s1>>>(...);
    // kernel2<<<...,s2>>>(...);
    // - padding
    float *dev_padded;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    // -- timer on
    hipEvent_t pad_start, pad_stop;
	hipEventCreate(&pad_start);
	hipEventCreate(&pad_stop); 
    float pad_time = 0;
    hipEventRecord(pad_start, 0);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_input, dev_padded, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    // -- timer off
    hipDeviceSynchronize();
    hipEventRecord(pad_stop, 0);
	hipEventSynchronize(pad_stop);
    hipEventElapsedTime(&pad_time, pad_start, pad_stop);
    // - convolution
    // -- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);
    // -- perform conv
    float *dev_conv;
    hipMalloc((void**)&dev_conv, height * width * out_channels * sizeof(float));
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(width/32+1, height/32+1, out_channels);
    // -- timer on
    hipEvent_t conv_start, conv_stop;
	hipEventCreate(&conv_start);
	hipEventCreate(&conv_stop); 
    float conv_time = 0;
    hipEventRecord(conv_start, 0);
    SharedConv2DReLU<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_conv,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // -- relu
    //ReLU<<<conv_grid_size, conv_block_size>>>(dev_conv, out_channels, height, width);
    // -- timer off
    hipDeviceSynchronize();
    hipEventRecord(conv_stop, 0);
	hipEventSynchronize(conv_stop);
    hipEventElapsedTime(&conv_time, conv_start, conv_stop);
    // - maxpool
    dim3 pool_block_size(32, 32, 1);
    dim3 pool_grid_size((width/2)/32+1, (height/2)/32+1, out_channels);
    // -- timer on
    hipEvent_t pool_start, pool_stop;
	hipEventCreate(&pool_start);
	hipEventCreate(&pool_stop); 
    float pool_time = 0;
    hipEventRecord(pool_start, 0);
    MaxPool2D<<<pool_grid_size, pool_block_size>>>
    (
        dev_conv, dev_output, out_channels, height, width, 2,2,2,2
    );
    // -- timer off
    hipDeviceSynchronize();
    hipEventRecord(pool_stop, 0);
	hipEventSynchronize(pool_stop);
    hipEventElapsedTime(&pool_time, pool_start, pool_stop);

    hipEventDestroy(pad_start);
    hipEventDestroy(pad_stop);
    hipEventDestroy(conv_start);
    hipEventDestroy(conv_stop);
    hipEventDestroy(pool_start);
    hipEventDestroy(pool_stop);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);
    hipFree(dev_conv);

    return pad_time + conv_time + pool_time;
}


float decoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    float *dev_upsampled;
    int up_height = height + height-1, up_width = width + width-1;
    hipMalloc((void**)&dev_upsampled, up_height * up_width * in_channels * sizeof(float));
    dim3 up_block_size(32, 32, 1);
    dim3 up_grid_size(up_width/32+1, up_height/32+1, in_channels);
    hipEvent_t up_start, up_stop;
	hipEventCreate(&up_start);
	hipEventCreate(&up_stop); 
    float up_time = 0;
    hipEventRecord(up_start, 0);
    ChessUpsample2D<<<up_grid_size, up_block_size>>>(dev_input, dev_upsampled, in_channels, height, width);
    hipDeviceSynchronize();
    hipEventRecord(up_stop, 0);
	hipEventSynchronize(up_stop);
    hipEventElapsedTime(&up_time, up_start, up_stop);

    float *dev_padded;
    int up_pad = 1, down_pad = 2, left_pad = 1, right_pad = 2;
    int padded_height= up_height+up_pad+down_pad, padded_width = up_width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    hipEvent_t pad_start, pad_stop;
	hipEventCreate(&pad_start);
	hipEventCreate(&pad_stop); 
    float pad_time = 0;
    hipEventRecord(pad_start, 0);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_upsampled, dev_padded, in_channels, up_height, up_width, up_pad, down_pad, left_pad, right_pad);
    hipDeviceSynchronize();
    hipEventRecord(pad_stop, 0);
	hipEventSynchronize(pad_stop);
    hipEventElapsedTime(&pad_time, pad_start, pad_stop);
    // -- convolve
    // ---- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    dim3 weight_block_size(32, 32, 1);
    dim3 weight_grid_size(3/32+1, 3/32+1, out_channels*in_channels);
    float *dev_conv_weight_flip;
    hipMalloc((void**)&dev_conv_weight_flip, out_channels*in_channels*3*3 * sizeof(float));
    hipEvent_t trans_start, trans_stop;
	hipEventCreate(&trans_start);
	hipEventCreate(&trans_stop); 
    float trans_time = 0;
    hipEventRecord(trans_start, 0);
    TransposeKernel<<<weight_grid_size, weight_block_size>>>(dev_conv_weight, dev_conv_weight_flip, out_channels, in_channels, 3, 3);
    hipDeviceSynchronize();
    hipEventRecord(trans_stop, 0);
	hipEventSynchronize(trans_stop);
    hipEventElapsedTime(&trans_time, trans_start, trans_stop);
    // --- load bias
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);

    // ---- perform conv
    int conv_height = height*2, conv_width = width*2;
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(conv_width/32+1, conv_height/32+1, out_channels);
    hipEvent_t conv_start, conv_stop;
	hipEventCreate(&conv_start);
	hipEventCreate(&conv_stop); 
    float conv_time = 0;
    hipEventRecord(conv_start, 0);
    SharedConv2DReLU<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight_flip, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // relu
    //ReLU<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, conv_height, conv_width);
    hipDeviceSynchronize();
    hipEventRecord(conv_stop, 0);
	hipEventSynchronize(conv_stop);
    hipEventElapsedTime(&conv_time, conv_start, conv_stop);

    hipEventDestroy(up_start);
    hipEventDestroy(up_stop);
    hipEventDestroy(pad_start);
    hipEventDestroy(pad_stop);
    hipEventDestroy(trans_start);
    hipEventDestroy(trans_stop);
    hipEventDestroy(conv_start);
    hipEventDestroy(conv_stop);
    hipFree(dev_upsampled);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);
    hipFree(dev_conv_weight_flip);

    return up_time + pad_time + trans_time + conv_time;
}


float refine_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    float *dev_padded;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    hipEvent_t pad_start, pad_stop;
	hipEventCreate(&pad_start);
	hipEventCreate(&pad_stop); 
    float pad_time = 0;
    hipEventRecord(pad_start, 0);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_input, dev_padded, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    hipDeviceSynchronize();
    hipEventRecord(pad_stop, 0);
	hipEventSynchronize(pad_stop);
    hipEventElapsedTime(&pad_time, pad_start, pad_stop);

    // -- convolve
    // ---- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);
    // ---- perform conv
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(width/32+1, height/32+1, out_channels);
    hipEvent_t conv_start, conv_stop;
	hipEventCreate(&conv_start);
	hipEventCreate(&conv_stop); 
    float conv_time = 0;
    hipEventRecord(conv_start, 0);
    SharedConv2DSigmoid<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    //Sigmoid<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, height, width);
    hipDeviceSynchronize();
    hipEventRecord(conv_stop, 0);
	hipEventSynchronize(conv_stop);
    hipEventElapsedTime(&conv_time, conv_start, conv_stop);

    hipEventDestroy(conv_start);
    hipEventDestroy(conv_stop);
    hipEventDestroy(pad_start);
    hipEventDestroy(pad_stop);
    hipFree(dev_padded);
    hipFree(dev_conv_weight);
    hipFree(dev_conv_bias);

    return conv_time+pad_time;
}