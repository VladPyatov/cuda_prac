#include "cnn_autoencoder.cuh"
#include "layers.cuh"
#include "utils.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void denoise(const uint8_t* input, uint8_t* result, std::string weights_path, int height, int width)
{
    hipEvent_t global_start, global_stop;
    hipEventCreate(&global_start);
	hipEventCreate(&global_stop);
    float time;
    hipEventRecord(global_start, 0);

    fs::path weights_path_object = fs::path(weights_path);

    // allocate memory for encoder weights
    float enc_conv1_weight[32*1*3*3];
    float enc_conv1_bias[32];
    float enc_conv2_weight[32*32*3*3];
    float enc_conv2_bias[32];
    // allocate memory for decoder weights
    float dec_tconv1_weight[32*32*3*3];
    float dec_tconv1_bias[32];
    float dec_tconv2_weight[32*32*3*3];
    float dec_tconv2_bias[32];
    float dec_conv1_weight[1*32*3*3];
    float dec_conv1_bias[1];

    // load encoder weights
    load_weights(enc_conv1_weight, weights_path_object/"enc_conv1_weight.bin", 32*1*3*3);
    load_weights(enc_conv1_bias, weights_path_object/"enc_conv1_bias.bin", 32);
    load_weights(enc_conv2_weight, weights_path_object/"enc_conv2_weight.bin", 32*32*3*3);
    load_weights(enc_conv2_bias, weights_path_object/"enc_conv2_bias.bin", 32);
    // load decoder weights
    load_weights(dec_tconv1_weight, weights_path_object/"dec_tconv1_weight.bin", 32*32*3*3);
    load_weights(dec_tconv1_bias, weights_path_object/"dec_tconv1_bias.bin", 32);
    load_weights(dec_tconv2_weight, weights_path_object/"dec_tconv2_weight.bin", 32*32*3*3);
    load_weights(dec_tconv2_bias, weights_path_object/"dec_tconv2_bias.bin", 32);
    load_weights(dec_conv1_weight, weights_path_object/"dec_conv1_weight.bin", 32*32*3*3);
    load_weights(dec_conv1_bias, weights_path_object/"dec_conv1_bias.bin", 1);

    for(int i=0; i<32; i++)
    {
        std::cout<< enc_conv1_bias[i] << std::endl;
    }
    
    // preprocess image: uint8 [0, 255] -> float [0, 1] 
    uint8_t *dev_uint_input;
    float *dev_input;
    hipMalloc((void**)&dev_uint_input, height *width * sizeof(uint8_t));
    hipMalloc((void**)&dev_input, height *width * sizeof(float));
    hipMemcpy(dev_uint_input, input, height * width * sizeof(uint8_t), hipMemcpyHostToDevice);
    dim3 img_block_size(32, 32);
    dim3 img_grid_size(width/32+1, height/32+1);
    img2float<<<img_grid_size, img_block_size>>>(dev_uint_input, dev_input, height, width);
    // preprocessing done

    //*****ENCODER*****
    // - layer 1
    int out_channels = 32;
    int in_channels = 1;
    float *dev_pool_layer1;
    int layer1_out_height = height/2, layer1_out_width = width/2;
    hipMalloc((void**)&dev_pool_layer1, layer1_out_height * layer1_out_width * out_channels * sizeof(float));
    encoder_layer(dev_input, dev_pool_layer1, enc_conv1_weight, enc_conv1_bias, in_channels, out_channels, height, width);
    // - layer 2
    out_channels = 32;
    in_channels = 32;
    float *dev_pool_layer2;
    int layer2_out_height = layer1_out_height/2, layer2_out_width = layer1_out_width/2;
    hipMalloc((void**)&dev_pool_layer2, layer2_out_height * layer2_out_width * out_channels * sizeof(float));
    encoder_layer(dev_pool_layer1, dev_pool_layer2, enc_conv2_weight, enc_conv2_bias, in_channels, out_channels, layer1_out_height, layer1_out_width);
    //*****DECODER*****
    // - layer 3
    out_channels = 32;
    in_channels = 32;
    float *dev_trans_layer3;
    int layer3_out_height = layer1_out_height, layer3_out_width = layer1_out_width;
    hipMalloc((void**)&dev_trans_layer3, layer3_out_height * layer3_out_width * out_channels * sizeof(float));
    decoder_layer(dev_pool_layer2, dev_trans_layer3, dec_tconv1_weight, dec_tconv1_bias, in_channels, out_channels, layer2_out_height, layer2_out_width);
    // - layer 4
    out_channels = 32;
    in_channels = 32;
    float *dev_trans_layer4;
    int layer4_out_height = height, layer4_out_width = width;
    hipMalloc((void**)&dev_trans_layer4, layer4_out_height * layer4_out_width * out_channels * sizeof(float));
    decoder_layer(dev_trans_layer3, dev_trans_layer4, dec_tconv2_weight, dec_tconv2_bias, in_channels, out_channels, layer3_out_height, layer3_out_width);
    // - layer 5
    out_channels = 1;
    in_channels = 32;
    float *dev_result;
    hipMalloc((void**)&dev_result, height * width * out_channels * sizeof(float));
    refine_layer(dev_trans_layer4, dev_result, dec_conv1_weight, dec_conv1_bias, in_channels, out_channels, layer4_out_height, layer4_out_width);

    // float pool_result[1*28*28];
    // hipMemcpy(pool_result, dev_result, 1*28*28 * sizeof(float), hipMemcpyDeviceToHost);
    // print_array(pool_result, height, width, 1);
    



    img2uint<<<img_grid_size, img_block_size>>>(dev_result, dev_uint_input, height, width);
    hipMemcpy(result, dev_uint_input, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipEventRecord(global_stop, 0);
	hipEventSynchronize(global_stop);
    hipEventElapsedTime(&time, global_start, global_stop);
    printf("time=%f\n",time);

    // // begin experiments
    // float array[] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27};
    // int array_h = 3;
    // int array_w = 3;
    // int array_c = 3;
    // int up_pad = 1;
    // int down_pad = 1;
    // int left_pad = 1;
    // int right_pad = 1;
    // float padded_array[(array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c] = {0};
    // print_array(array, array_h, array_w, array_c);
    // //print_array(padded_array, array_h, array_w, array_c);

    // float *dev_array;
    // float *dev_padded_array;
    
    // hipMalloc((void**)&dev_array, array_h * array_w * array_c * sizeof(float));
    // hipMalloc((void**)&dev_padded_array, (array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c * sizeof(float));

    // hipMemcpy(dev_array, array, array_h * array_w * array_c * sizeof(float), hipMemcpyHostToDevice);
    // const dim3 exp_block_size(32, 32, 1);
    // const dim3 exp_grid_size((array_w + left_pad + right_pad)/32+1, (array_h + up_pad + down_pad)/32+1, array_c);
    // ZeroPad2D<<<exp_grid_size, exp_block_size>>>(dev_array, dev_padded_array, array_c, array_h, array_w, up_pad, down_pad, left_pad, right_pad);
    // hipMemcpy(padded_array, dev_padded_array, (array_h + up_pad + down_pad) * (array_w + left_pad + right_pad) * array_c * sizeof(float), hipMemcpyDeviceToHost);
    // // convolution
    // int out_channels = 2;
    // float weight[] = {
    //     1,2,0,-1,1,3,2,1,0, 2,1,2,0,1,2,0,0,1, 1,1,0,2,1,2,0,1,0,
    //     1,1,2,0,3,0,1,2,3, -1,2,1,0,1,2,0,1,0, 1,1,0,2,1,0,2,2,2,
    // };
    // float bias[] = {1, 2};

    // float *dev_convolved_array;
    // float *dev_weight;
    // float *dev_bias;
    
    // hipMalloc((void**)&dev_convolved_array, array_h * array_w * out_channels * sizeof(float));
    // hipMalloc((void**)&dev_weight, 3 * 3 * 6 * sizeof(float));
    // hipMalloc((void**)&dev_bias, 2 * sizeof(float));
    // hipMemcpy(dev_weight, weight, 3 * 3 * 6 * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(dev_bias, bias, 2 * sizeof(float), hipMemcpyHostToDevice);
    // dim3 conv_block_size(32, 32, 1);
    // dim3 conv_grid_size(array_w/32+1, array_h/32+1, out_channels);
    // Conv2D<<<conv_grid_size, conv_block_size>>>(dev_padded_array, dev_weight, dev_bias, dev_convolved_array, 3, 2, array_h + up_pad + down_pad, array_w + left_pad + right_pad, 3, 3);
    // hipMemcpy(array, dev_convolved_array, array_h * array_w * out_channels * sizeof(float), hipMemcpyDeviceToHost);
    // // MaxPool2D
    // float *dev_pooled_array;
    // float pooled_array[2*2*3];
    // hipMalloc((void**)&dev_pooled_array, ((array_w + left_pad + right_pad)/2) * ((array_h + up_pad + down_pad)/2) * array_c * sizeof(float));
    // dim3 pool_block_size(32, 32, 1);
    // dim3 pool_grid_size((array_w + left_pad + right_pad)/2/32+1, (array_h + up_pad + down_pad)/2/32+1, array_c);
    // MaxPool2D<<<pool_grid_size, pool_block_size>>>(dev_padded_array, dev_pooled_array, array_c, array_h + up_pad + down_pad, array_w + left_pad + right_pad, 2,2,2,2);
    // hipMemcpy(pooled_array, dev_pooled_array, ((array_w + left_pad + right_pad)/2) * ((array_h + up_pad + down_pad)/2) * array_c * sizeof(float), hipMemcpyDeviceToHost);
    // //Upsample
    // // MaxPool2D
    // float *dev_up_array;
    // float up_array[2*5*5];
    // hipMalloc((void**)&dev_up_array, 5*5*2* sizeof(float));
    // dim3 up_block_size(32, 32, 1);
    // dim3 up_grid_size(5/32+1, 5/32+1, 2);
    // ChessUpsample2D<<<up_grid_size, up_block_size>>>(dev_convolved_array, dev_up_array, 2, 3, 3);
    // hipMemcpy(up_array, dev_up_array, 2*5*5 * sizeof(float), hipMemcpyDeviceToHost);
    // // Flip Weight
    // printf("####WEIGHT####\n");
    // print_array(weight, 3, 3, 3*2);
    // dim3 weight_block_size(32, 32, 1);
    // dim3 weight_grid_size(3/32+1, 3/32+1, 2);
    // FlipWeight2D<<<weight_grid_size, weight_block_size>>>(dev_weight, 2, 3, 3, 3);
    // hipMemcpy(weight, dev_weight, 3*3*2*3 * sizeof(float), hipMemcpyDeviceToHost);
    // printf("####F_WEIGHT####\n");
    // print_array(weight, 3, 3, 3*2);
    // //hipFree(dev_array);
    // //hipFree(dev_padded_array);
    
    // print_array(padded_array, array_h + up_pad + down_pad, array_w + left_pad + right_pad, array_c);
    // print_array(array, array_h, array_w, out_channels);
    // print_array(pooled_array, (array_h + up_pad + down_pad)/2, (array_w + left_pad + right_pad)/2, array_c);
    // print_array(up_array, 5, 5, 2);
    // // end experiment

    hipFree(dev_input);
    hipFree(dev_uint_input);
}

void encoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    // hipStream_t s1, s2;
    // cudaStreaCreate(&s1);
    // hipStreamCreate(&s2);
    // kernel1<<<...,s1>>>(...);
    // kernel2<<<...,s2>>>(...);
    float *dev_padded_layer1;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded_layer1, padded_height * padded_width * in_channels * sizeof(float));
    dim3 enc1_pad_block_size(32, 32, 1);
    dim3 enc1_pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<enc1_pad_grid_size, enc1_pad_block_size>>>(dev_input, dev_padded_layer1, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    // -- convolve
    // ---- load weights
    float *dev_enc_conv1_weight;
    hipMalloc((void**)&dev_enc_conv1_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_enc_conv1_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    float *dev_enc_conv1_bias;
    hipMalloc((void**)&dev_enc_conv1_bias, out_channels * sizeof(float));
    hipMemcpy(dev_enc_conv1_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);
    // ---- perform conv
    float *dev_conv_layer1;
    hipMalloc((void**)&dev_conv_layer1, height * width * out_channels * sizeof(float));
    dim3 enc1_conv_block_size(32, 32, 1);
    dim3 enc1_conv_grid_size(width/32+1, height/32+1, out_channels);
    Conv2D<<<enc1_conv_grid_size, enc1_conv_block_size>>>
    (
        dev_padded_layer1, dev_enc_conv1_weight, dev_enc_conv1_bias, dev_conv_layer1,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // relu
    ReLU<<<enc1_conv_grid_size, enc1_conv_block_size>>>(dev_conv_layer1, out_channels, height, width);
    // -- maxpool
    dim3 enc1_pool_block_size(32, 32, 1);
    dim3 enc1_pool_grid_size((width/2)/32+1, (height/2)/32+1, out_channels);
    MaxPool2D<<<enc1_pool_grid_size, enc1_pool_block_size>>>
    (
        dev_conv_layer1, dev_output, out_channels, height, width, 2,2,2,2
    );
}

void decoder_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    float *dev_upsampled;
    int up_height = height + height-1, up_width = width + width-1;
    hipMalloc((void**)&dev_upsampled, up_height * up_width * in_channels * sizeof(float));
    dim3 up_block_size(32, 32, 1);
    dim3 up_grid_size(up_width/32+1, up_height/32+1, in_channels);
    ChessUpsample2D<<<up_grid_size, up_block_size>>>(dev_input, dev_upsampled, in_channels, height, width);


    float *dev_padded;
    int up_pad = 1, down_pad = 2, left_pad = 1, right_pad = 2;
    int padded_height= up_height+up_pad+down_pad, padded_width = up_width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_upsampled, dev_padded, in_channels, up_height, up_width, up_pad, down_pad, left_pad, right_pad);
    // -- convolve
    // ---- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    dim3 weight_block_size(32, 32, 1);
    dim3 weight_grid_size(3/32+1, 3/32+1, out_channels);
    FlipWeight2D<<<weight_grid_size, weight_block_size>>>(dev_conv_weight, out_channels, in_channels, 3, 3);
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);

    // ---- perform conv
    int conv_height = height*2, conv_width = width*2;
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(conv_width/32+1, conv_height/32+1, out_channels);
    Conv2D<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // relu
    ReLU<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, conv_height, conv_width);
}


void decoder_layer_memory(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    float *dev_upsampled;
    int up_height = height + height-1, up_width = width + width-1;
    hipMalloc((void**)&dev_upsampled, up_height * up_width * in_channels * sizeof(float));
    dim3 up_block_size(32, 32, 1);
    dim3 up_grid_size(up_width/32+1, up_height/32+1, in_channels);
    ChessUpsample2D<<<up_grid_size, up_block_size>>>(dev_input, dev_upsampled, in_channels, height, width);


    float *dev_padded;
    int up_pad = 1, down_pad = 2, left_pad = 1, right_pad = 2;
    int padded_height= up_height+up_pad+down_pad, padded_width = up_width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_upsampled, dev_padded, in_channels, up_height, up_width, up_pad, down_pad, left_pad, right_pad);
    // -- convolve
    // ---- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    dim3 weight_block_size(32, 32, 1);
    dim3 weight_grid_size(3/32+1, 3/32+1, out_channels*in_channels);
    float *dev_conv_weight_flip;
    hipMalloc((void**)&dev_conv_weight_flip, out_channels*in_channels*3*3 * sizeof(float));
    TransposeKernel<<<weight_grid_size, weight_block_size>>>(dev_conv_weight, dev_conv_weight_flip, out_channels, in_channels, 3, 3);
    
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);

    // ---- perform conv
    int conv_height = height*2, conv_width = width*2;
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(conv_width/32+1, conv_height/32+1, out_channels);
    Conv2D<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight_flip, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    // relu
    ReLU<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, conv_height, conv_width);
}

void refine_layer(float *dev_input, float *dev_output, float *weight, float *bias, int in_channels, int out_channels, int height, int width)
{
    float *dev_padded;
    int up_pad = 1, down_pad = 1, left_pad = 1, right_pad = 1;
    int padded_height= height+up_pad+down_pad, padded_width = width+left_pad+right_pad;
    hipMalloc((void**)&dev_padded, padded_height * padded_width * in_channels * sizeof(float));
    dim3 pad_block_size(32, 32, 1);
    dim3 pad_grid_size(padded_width/32+1, padded_height/32+1, in_channels);
    ZeroPad2D<<<pad_grid_size, pad_block_size>>>(dev_input, dev_padded, in_channels, height, width, up_pad, down_pad, left_pad, right_pad);
    // -- convolve
    // ---- load weights
    float *dev_conv_weight;
    hipMalloc((void**)&dev_conv_weight, out_channels*in_channels*3*3 * sizeof(float));
    hipMemcpy(dev_conv_weight, weight, out_channels*in_channels*3*3 * sizeof(float), hipMemcpyHostToDevice);
    float *dev_conv_bias;
    hipMalloc((void**)&dev_conv_bias, out_channels * sizeof(float));
    hipMemcpy(dev_conv_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);
    // ---- perform conv
    dim3 conv_block_size(32, 32, 1);
    dim3 conv_grid_size(width/32+1, height/32+1, out_channels);
    Conv2D<<<conv_grid_size, conv_block_size>>>
    (
        dev_padded, dev_conv_weight, dev_conv_bias, dev_output,
        in_channels, out_channels, padded_height, padded_width, 3, 3
    );
    Sigmoid<<<conv_grid_size, conv_block_size>>>(dev_output, out_channels, height, width);

}